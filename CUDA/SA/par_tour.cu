#include "hip/hip_runtime.h"
#include "tour.h"
#include "config.h"
#include "town.h"
#include "pools.h"
#include <stdlib.h>

#define NBLOCKS 5
#define THREADS_PER_BLOCK (NUM_VERTEXES/NBLOCKS)

Tour * tour_new(TownList *towns, town_index_t num_vertexes){
    Tour * newtour = (Tour*) malloc(sizeof(Tour));
    newtour->tour = (town_index_t *) malloc(sizeof(town_index_t)*num_vertexes);
    newtour->town_list = towns;
    newtour->num_vertexes = num_vertexes;
    return newtour;
}

void tour_randomize(Tour * tour) {
    town_index_t i, num_vertexes=tour->num_vertexes;
    TownPool *tp = tp_new(num_vertexes);
    
    for (i=0; i<num_vertexes; i++){
        tour->tour[i] = tp_random_town(tp);
    }

    tp_destroy(tp);
}

Tour * tour_destroy(Tour * tour){
    free(tour->tour);
    free(tour);
    return tour = NULL;
}

static Tour * d_tour_new(Tour *h_t){
    Tour *d_t;
    hipMalloc((void**)&d_t, sizeof(Tour));
    hipMalloc((void**)&d_t->tour, NUM_VERTEXES*sizeof(town_index_t));
    d_t->town_list = h_t->town_list;
    return d_t;
}

__device__ double tl_distance(TownList * t_list, town_index_t i1, town_index_t i2){
    Town *t1 = &t_list->list[i1], *t2 = &t_list->list[i2];
    double dx = t2->x - t1->x;
    double dy = t2->y - t1->y;
    return sqrt(dx*dx + dy*dy);
}

static __global__ void calc_len(Tour * t, double *part_len){
    __shared__ double s_c[THREADS_PER_BLOCK];

	int tx = threadIdx.x;
	unsigned i = blockIdx.x * blockDim.x + threadIdx.x;

	// Initialize shared memory.
	s_c[tx] = tl_distance(t->town_list, i, (i+1)%NUM_VERTEXES);
	__syncthreads();

	// Reduce in shared memory.
	// Good version (v2)
	int stride = THREADS_PER_BLOCK >> 1 ;
	for (  ; tx < stride; stride >>= 1 )	
	{		
	s_c[tx] += s_c[tx + stride];
	__syncthreads();
	}

	// Thread zero writes the result to global memory.
	if ( tx < 1 )
		part_len[ blockIdx.x ] = s_c[ 0 ] ; // v1
}

double tour_set_len(Tour * tour){
    Tour *d_tour;
    double len = 0, h_part_len[NBLOCKS], *d_part_len;
    int i;

    d_tour = d_tour_new(tour);
    hipMalloc((void**)&d_part_len, NBLOCKS*sizeof(double));

    calc_len<<<NBLOCKS, THREADS_PER_BLOCK>>>(d_tour, d_part_len);
    hipMemcpy(h_part_len, d_part_len, NBLOCKS*sizeof(double), hipMemcpyDeviceToHost);

    for (i=0; i<NBLOCKS; i++)
        len += h_part_len[i];

    hipFree(d_part_len);

    tour->length = len;
    return len;
}

void tour_mutate(Tour *new_tour, Tour *old_tour) {
    town_index_t i;
    TownPool *tp;

    tp = tp_new(NUM_VERTEXES);

    for (i=0; i<NUM_VERTEXES; i++){
        new_tour->tour[i] = old_tour->tour[i];
    }

    {
        town_index_t i1, i2;
        i1 = (town_index_t)tp_random_town(tp);
        i2 = (town_index_t)tp_random_town(tp);
        new_tour->tour[i1] = old_tour->tour[i2];
        new_tour->tour[i2] = old_tour->tour[i1];
    }

    tp_destroy(tp);
}
