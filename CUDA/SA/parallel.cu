#include "hip/hip_runtime.h"
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

#include "pools.h"
#include "config.h"
#include "town.h"
#include "tour.h"


/********************** POOLS.CU ***************************/

struct townpool{
    town_index_t num_indexes;
    town_index_t n_unused;
    unsigned char *arr;
};

TownPool * tp_new(town_index_t num_indexes){
    size_t size = (size_t) ceil(num_indexes/8.0);
    TownPool *tp =  (TownPool*) malloc(sizeof(TownPool));
    tp->arr = (unsigned char *) malloc(size);
    memset(tp->arr, 0xff, size);

    tp->n_unused    = num_indexes;
    tp->num_indexes = num_indexes;

    return tp; 
}

TownPool * tp_destroy(TownPool *tp){
    free(tp->arr);
    free(tp);
    return tp = NULL;
}

int tp_usable(TownPool *tp, town_index_t i){
    return ((unsigned char)1 << i%8 & tp->arr[i/8]) != 0;
}

int tp_unusable(TownPool *tp, town_index_t i){
    return ((unsigned char)1 << i%8 & tp->arr[i/8]) == 0;
}

void tp_remove(TownPool *tp, town_index_t i){
    unsigned char mask = (unsigned char)1 << i%8, current = tp->arr[i/8];
    tp->arr[i/8] = current & mask ? current ^ mask : current;
}

void tp_reset(TownPool *tp){
    memset(tp->arr, 0xff, (size_t)ceil(tp->num_indexes/8.0));
}

#define random_town() (rand() % NUM_VERTEXES)
town_index_t tp_random_town(TownPool *tp){
    town_index_t index = -1;
    if (tp->n_unused > 0){
        do {
            index = ((town_index_t)rand()) % tp->num_indexes;
        } while(tp_unusable(tp, index));
        tp_remove(tp, index);
        tp->n_unused--;
    }
    return index;
}


/********************** TOWN.CU ***************************/


TownList * tl_new(Config config){
    TownList * newt = (TownList *) malloc(sizeof(TownList));
    newt->list = (Town*) malloc(NUM_VERTEXES * sizeof(Town));
    newt->config = config;
    return newt;
}

#define RANDOM_COORD(min, max) (coord_t)(rand() % (max - min) + min);
static void add_random_town(TownList * t_list, town_index_t i){
    t_list->list[i].x = RANDOM_COORD(0, t_list->config.grid_size);
    t_list->list[i].y = RANDOM_COORD(0, t_list->config.grid_size);
}

void tl_randomize(TownList *tl){
    town_index_t i;
    for (i=0; i < NUM_VERTEXES; i++)
        add_random_town(tl, i);
}

TownList * tl_destroy(TownList * t_list){
    free(t_list->list);
    free(t_list);
    return t_list = NULL;
}


/********************** TOUR.CU ***************************/


#define NBLOCKS 5
#define THREADS_PER_BLOCK (NUM_VERTEXES/NBLOCKS)

Tour * tour_new(TownList *towns, town_index_t num_vertexes){
    Tour * newtour = (Tour*) malloc(sizeof(Tour));
    newtour->tour = (town_index_t *) malloc(sizeof(town_index_t)*num_vertexes);
    newtour->town_list = towns;
    newtour->num_vertexes = num_vertexes;
    return newtour;
}

void tour_randomize(Tour * tour) {
    town_index_t i, num_vertexes=tour->num_vertexes;
    TownPool *tp = tp_new(num_vertexes);
    
    for (i=0; i<num_vertexes; i++){
        tour->tour[i] = tp_random_town(tp);
    }

    tp_destroy(tp);
}

Tour * tour_destroy(Tour * tour){
    free(tour->tour);
    free(tour);
    return tour = NULL;
}

static town_index_t * d_tour_new(Tour *h_t){
    town_index_t *d_t;
    hipMalloc((void**)&d_t, NUM_VERTEXES*sizeof(town_index_t));
    hipMemcpy(d_t, h_t->tour, NUM_VERTEXES*sizeof(town_index_t), hipMemcpyHostToDevice);
    return d_t;
}

__device__ double tl_distance(TownList * t_list, town_index_t i1, town_index_t i2){
    Town *t1 = &t_list->list[i1], *t2 = &t_list->list[i2];
    double dx = t2->x - t1->x;
    double dy = t2->y - t1->y;
    return sqrt(dx*dx + dy*dy);
}

static __global__ void calc_len(TownList *tl, town_index_t *t, double *part_len){
    __shared__ double s_c[THREADS_PER_BLOCK];

	int tx = threadIdx.x;
	unsigned i = blockIdx.x * blockDim.x + threadIdx.x;

	// Initialize shared memory.
	s_c[tx] = tl_distance(tl, t[i], t[(i+1)%NUM_VERTEXES]);
	__syncthreads();

	// Reduce in shared memory.
	// Good version (v2)
	int stride = THREADS_PER_BLOCK >> 1 ;
	for (  ; tx < stride; stride >>= 1 )	
	{		
	s_c[tx] += s_c[tx + stride];
	__syncthreads();
	}

	// Thread zero writes the result to global memory.
	if ( tx < 1 )
		part_len[ blockIdx.x ] = s_c[ 0 ] ; // v1
}

double tour_set_len(Tour * tour){
    town_index_t *d_tour;
    double len = 0, h_part_len[NBLOCKS], *d_part_len;
    int i;

    d_tour = d_tour_new(tour);
    hipMalloc((void**)&d_part_len, NBLOCKS*sizeof(double));

    calc_len<<<NBLOCKS, THREADS_PER_BLOCK>>>(tour->town_list, d_tour, d_part_len);
    hipMemcpy(h_part_len, d_part_len, NBLOCKS*sizeof(double), hipMemcpyDeviceToHost);

    for (i=0; i<NBLOCKS; i++)
        len += h_part_len[i];

    hipFree(d_part_len);
    hipFree(d_tour);

    tour->length = len;
    return len;
}

void tour_mutate(Tour *new_tour, Tour *old_tour) {
    town_index_t i;
    TownPool *tp;

    tp = tp_new(NUM_VERTEXES);

    for (i=0; i<NUM_VERTEXES; i++){
        new_tour->tour[i] = old_tour->tour[i];
    }

    {
        town_index_t i1, i2;
        i1 = (town_index_t)tp_random_town(tp);
        i2 = (town_index_t)tp_random_town(tp);
        new_tour->tour[i1] = old_tour->tour[i2];
        new_tour->tour[i2] = old_tour->tour[i1];
    }

    tp_destroy(tp);
}

/********************** MAIN.CU ***************************/

#define cron_begin clock
#define cron_end(start) ((clock()-start)/CLOCKS_PER_SEC)

int should_replace(Tour *newt, Tour *old, double temperature){
    double dl = newt->length - old->length;
    return dl<0 ? 1 : rand()/RAND_MAX < exp(-dl/temperature);
}


int main(const int argc, const char *argv[]){
    Config state = {
        GRID_SIZE,
        NUM_VERTEXES,
        ALPHA,
        EPSILON,
        TEMPERATURE
    };

    TownList * h_towns, *d_towns;
    Tour *old_tour, *new_tour;
    int i = 0;
    double start;

    //puts("Executing sequential version");

    srand((int)time(NULL));

    start = cron_begin();

    h_towns = tl_new(state);
    tl_randomize(h_towns);
    hipMalloc((void**)&d_towns, NUM_VERTEXES*sizeof(Town));
    hipMemcpy(d_towns, h_towns, NUM_VERTEXES*sizeof(Town), hipMemcpyHostToDevice);
    tl_destroy(h_towns);

    old_tour = tour_new(d_towns, NUM_VERTEXES); 
    tour_randomize(old_tour);
    tour_set_len(old_tour);

    new_tour = tour_new(d_towns, NUM_VERTEXES);
    while (state.temperature > state.epsilon){
        i++;
        tour_mutate(new_tour, old_tour);
        tour_set_len(new_tour);
        if (should_replace(new_tour, old_tour, state.temperature)){
            Tour * dummy = old_tour;
            old_tour = new_tour;
            new_tour = dummy;
        }
        
        state.temperature *= state.alpha;
    }
    

    printf("Sequential time: %f\n", cron_end(start));
    //printf("After %d iterations, the best length: %f\n", i, old_tour->length);

    tour_destroy(old_tour);
    tour_destroy(new_tour);
    hipFree(d_towns);
    return 0;
}
