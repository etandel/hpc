#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "town.h"
#include "config.h"

TownList * tl_new(Config config){
    TownList * newt = (TownList *) malloc(sizeof(TownList));
    newt->list = (Town*) malloc(NUM_VERTEXES * sizeof(Town));
    newt->config = config;
    return newt;
}

#define RANDOM_COORD(min, max) (coord_t)(rand() % (max - min) + min);
static void add_random_town(TownList * t_list, town_index_t i){
    t_list->list[i].x = RANDOM_COORD(0, t_list->config.grid_size);
    t_list->list[i].y = RANDOM_COORD(0, t_list->config.grid_size);
}

void tl_randomize(TownList *tl){
    town_index_t i;
    for (i=0; i < NUM_VERTEXES; i++)
        add_random_town(tl, i);
}

TownList * tl_destroy(TownList * t_list){
    free(t_list->list);
    free(t_list);
    return t_list = NULL;
}

