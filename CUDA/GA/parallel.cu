#include "hip/hip_runtime.h"
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""

#include "config.h"
#include "pools.h"
#include "town.h"
#include "population.h"

Subject *d_subjs;

/******************** POOLS.C **********************/

struct genepool {
    unsigned char *pool;
    gene_t *seq;
    gene_t i;
    gene_t pool_size;
    gene_t seq_size;
};

GenePool gp_new(void){
    gene_t i;
    GenePool gp   = (GenePool) malloc(sizeof(struct genepool));
    gp->i         = 0;
    gp->seq_size  = NUM_VERTEXES;
    gp->seq       = (gene_t *) malloc(gp->seq_size*sizeof(gene_t));

    gp->pool_size = (size_t)ceil(gp->seq_size/8.0);
    gp->pool      = (unsigned char *) malloc(gp->pool_size);
    memset(gp->pool, 0xff, gp->pool_size);

    for (i=0; i<gp->seq_size; i++)
        gp->seq[i] = i;

    gp_reset(gp);
    return gp; 
}

void gp_destroy(GenePool gp){
    free(gp->seq);
    free(gp->pool);
    free(gp);
}

int gp_usable(GenePool gp, gene_t g){
    return (((unsigned char)1 << g%8 & gp->pool[g/8]) != 0);
}

int gp_unusable(GenePool gp, gene_t g){
    return (((unsigned char)1 << g%8 & gp->pool[g/8]) == 0);
}


void gp_remove(GenePool gp, gene_t g){
    unsigned char mask = (unsigned char)1 << g%8, current = gp->pool[g/8];
    gp->pool[g/8] = current & mask ? current ^ mask : current;
}

void gp_reset(GenePool gp){
    int i;

    for (i=gp->seq_size-1; i>0; i--){
        gene_t j = rand() % (i+1);
        gene_t tmp = gp->seq[i];
        gp->seq[i] = gp->seq[j];
        gp->seq[j] = tmp;
    }
}

gene_t gp_get_random(GenePool gp){
    gene_t g = gp->seq[gp->i++];
    gp_remove(gp, g);
    return g;
}


/******************** TOWN.C **********************/


Town * tl_new(gene_t nvertex){
    Town * newt;
    nvertex = nvertex ? nvertex : NUM_VERTEXES;
    newt = (Town *) malloc(NUM_VERTEXES * sizeof(Town));
    return newt;
}

static coord_t random_coord(coord_t min, coord_t max){
    return (coord_t)(rand() % (max - min) + min);
}

static void add_random_town(Town * t_list, int i){
    t_list[i].x = random_coord(0, GRID_SIZE);
    t_list[i].y = random_coord(0, GRID_SIZE);
}

void tl_randomize(Town *town){
    gene_t i;
    for (i=0; i<NUM_VERTEXES; i++)
        add_random_town(town, i);
}

void tl_destroy(Town * t_list){
    free(t_list);
}

fit_t tl_distance(Town * t_list, gene_t g1, gene_t g2){
    Town *t1 = &t_list[g1], *t2 = &t_list[g2];
    fit_t dx = (fit_t) (t2->x - t1->x);
    fit_t dy = (fit_t) (t2->y - t1->y);
    return (fit_t) sqrt((double)(dx*dx + dy*dy));
}

/******************** POPULATION.C **********************/

#define random_gene() ((gene_t)(rand() % NUM_VERTEXES))

fit_t subj_tour_length(Subject * subj, Town *t_list){
    gene_t i = NUM_VERTEXES-1, *tour=subj->tour;
    fit_t total_len = 0;

    for (i=0; i<NUM_VERTEXES-1; i++){
        total_len += tl_distance(t_list, tour[i], tour[i+1]);
    }
    total_len += tl_distance(t_list, tour[i], tour[0]); //the return to the beginning

    return total_len;
}

fit_t calc_fitness(Subject * subj, Town *t_list){
    return -subj_tour_length(subj, t_list);
}

void subj_print_tour(Subject * subj, Town *t_list){
    gene_t i;
    for (i=0; i<NUM_VERTEXES; i++)
        printf("(%u, %u)\n", t_list[i].x, t_list[i].y);
}


/****** Beginning of random_new() and its auxiliary functions ******/

static void add_random_subj(Population * newp, subj_t i){
    gene_t gene_i;
    Subject * subj = newp->pop+i;
    GenePool gp = gp_new();

    for (gene_i=0; gene_i<NUM_VERTEXES; gene_i++)
        subj->tour[gene_i] = gp_get_random(gp);

    gp_destroy(gp);
}

Population *pop_new(Town *t_list){
    Population *newp = (Population*) malloc(sizeof(Population));
    newp->pop        = (Subject*) malloc(POP_SIZE*sizeof(Subject));
    newp->t_list     = t_list;
    return newp;
}

void pop_randomize(Population *newp){
    // randomizes population 
    subj_t i;

    //adds random subjs
    for (i=0; i<POP_SIZE; i++)
        add_random_subj(newp, i);
}

void pop_set_fit(Population *pop){
    subj_t i, fittest;
    fit_t max_fit=FIT_MIN;

    for (i=0; i<POP_SIZE; i++){
        // calcs and sets fitness related stuff
        Subject *subj = pop->pop+i;
        fit_t new_fit = calc_fitness(subj, pop->t_list);
        subj->fitness = new_fit;

        if (new_fit > max_fit){
            max_fit = new_fit;
            fittest = i;
        }
    }
    pop->max_fitness = max_fit;
    pop->fittest     = fittest;
}

/****** End of random_new() and its auxiliary functions ******/



/****** Beginning of next_generation() and its auxiliary functions ******/

//so that crossover point gives at least 1 gene for each parent
#define random_cross_point() (rand() % (NUM_VERTEXES-2) + 1)
static void crossover(gene_t *a_tour, gene_t *b_tour, gene_t *child_tour, GenePool gp ){
    gene_t i, cross_point = random_cross_point();
    gene_t holes[NUM_VERTEXES], nholes=0;

    //first part is equal to parent_a
    for (i=0; i<cross_point; i++){
        child_tour[i] = a_tour[i];
        gp_remove(gp, child_tour[i]);
    }

    //second part is equal to b unless the gene is unusable;
    //if so, keep a track of holes to be filled afterwards randomly
    for (; i<NUM_VERTEXES; i++){
        gene_t tmp = b_tour[i];
        if (gp_usable(gp, tmp)){
            child_tour[i] = tmp;
            gp_remove(gp, tmp);
        }
        else{
            //sets holes vec with index of hole
            holes[nholes++] = i;
        }
    }
            
    //hole filling with random usable genes
    for (i=0; i<nholes; i++){
        child_tour[holes[i]] = gp_get_random(gp);
        gp_remove(gp, child_tour[holes[i]]);
    }
}

static void mutate(Subject * subj){
    gene_t i2, i1=random_gene();
    gene_t tmp;
    do {
        i2 = random_gene();
    } while(i1 == i2);

    tmp = subj->tour[i1];
    subj->tour[i1] = subj->tour[i2];
    subj->tour[i1] = tmp;
}

#define MUST_CROSS  (rand()/(float)RAND_MAX<=CROSS_PROB)
#define MUST_MUTATE (rand()/(float)RAND_MAX<=MUTATION_PROB)
static Subject reproduce(Subject *parent_a, Subject *parent_b, Town * t_list){
    // returns possibly mutated child of a and b
    Subject new_subj;
    gene_t *tour = new_subj.tour , i;
    GenePool gp = gp_new();

    // if crossover shouldn't occur, child is equal to a
    if (MUST_CROSS)
        crossover(parent_a->tour, parent_b->tour, tour, gp);
    else
        for(i=0; i<NUM_VERTEXES; i++)
            new_subj.tour[i] = parent_a->tour[i];

    if (MUST_MUTATE)
        mutate(&new_subj);

    gp_destroy(gp);
    return new_subj;   
}

#define random_parent(old_pop) (old_pop->pop+(rand()%POP_SIZE))
void pop_reproduce(Population *newp, Population *oldp){
    //returns next generation
    subj_t i;
    Town *t_list = oldp->t_list;

    // generates, for every random pair of parents, 2 children
    for (i=0; i<POP_SIZE; i+=2){
        Subject *parent_a, *parent_b;

        parent_a = random_parent(oldp);
        do {
            parent_b = random_parent(oldp);
        } while (parent_a == parent_b);

        newp->pop[i]   = reproduce(parent_a, parent_b, t_list);
        newp->pop[i+1] = reproduce(parent_b, parent_a, t_list);
    }
}

/****** End of next_generation() and its auxiliary functions ******/



/****** Beginning of interface  ******/

//fit_t subj_tour_lenght(Population *pop, subj_t subj_i, Town *t_list){
//    /* length of the tour */
//}


void pop_destroy(Population * p){
    free(p->pop);
    free(p);
}


/******************** MAIN.C **********************/

#define cron_begin clock
#define cron_end(start) ((clock() - start) / CLOCKS_PER_SEC)

int main (const int argc, const char * argv[]){
    Town *t_list, *d_tlist;
    Population *parents, *children, *dummy;
    fit_t max_fitness = FIT_MIN;
    Subject fittest;
    int stag_count=0, iter=0;

    double start;
    
    //puts("Executing parallel version");

    srand((int)time(NULL)); //seed pseudo-rand generator
    start = cron_begin();

    // initialization
    t_list = tl_new(NUM_VERTEXES);
    tl_randomize(t_list);
    hipMalloc((void**)&d_tlist, NUM_VERTEXES*sizeof(Town));
    hipMemcpy(d_tlist, t_list, NUM_VERTEXES*sizeof(Town), hipMemcpyHostToDevice);
    tl_destroy(t_list);

    hipMalloc((void**)&d_subjs, POP_SIZE*sizeof(Subject));

    parents  = pop_new(d_tlist);
    pop_randomize(parents);
    pop_set_fit(parents);
    max_fitness = parents->max_fitness;
    fittest     = parents->pop[parents->fittest];

    children = pop_new(d_tlist);
    do {
        iter++;
        pop_reproduce(children, parents);
        pop_set_fit(children);
        if (children->max_fitness > max_fitness){
            fittest     = children->pop[children->fittest];
            max_fitness = children->max_fitness;
            stag_count  = 0;
            //printf("New max fitness: %.17f\n", max_fitness);
        }
        else
            stag_count++;

        dummy    = parents;
        parents  = children;
        children = dummy;
    } while(stag_count < STAG_COUNT);

    printf("Parallel iterations / time = %f\n", iter / cron_end(start));
    /*
        printf("Parallel iterations: %d ; parallel time: %f seconds.\n", iter, MPI_Wtime() - start);
        printf("length: %.17f\n", subj_tour_length(&fittest, t_list));
    // */
    hipFree(d_tlist);
    pop_destroy(parents);
    pop_destroy(children);
    hipFree(d_subjs);

    return 0;
}
